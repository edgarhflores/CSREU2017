#include "hip/hip_runtime.h"
#ifndef __STEREOVISIONAPP_DRIVER_CU
#define __STEREOVISIONAPP_DRIVER_CU

#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <cfloat>
#include <cmath>
#include "./tests/stereoVisionAppData.h"
#include "../codegenInput/UpdateApp.cuh"
#include "../codegenInput/EnergyApp.cuh"
#define ifL "/home/research/e.flores/StereoVision/tutorial-tsukuba-imL.pgm"
#define ifR "/home/research/e.flores/StereoVision/tutorial-tsukuba-imR.pgm"
#define of "/home/research/e.flores/StereoVision/tskuba-output.pgm"
#define convergence -0.001
#define numLabels 16
#define windowSize 5
#define lambda 20
#define truncate 2

int width, height, numPixels;

void importPGM(string filePath, unsigned char** pixels);
void setDC(int x, int y, int label, float val, float** DCArray);
float getDC(int x, int y, int label, float** DCArray);

void run_stereoVisionApp()
{
	unsigned char* leftPixels, * rightPixels;

	// import the images
	importPGM(ifL, &leftPixels);
	importPGM(ifR, &rightPixels);

	// calculate borders
	int leftBorder = (numLabels - 1) + (windowSize / 2);
	int rightBorder = width - (windowSize / 2) - 1;
	int topBorder = windowSize / 2;
	int bottomBorder = height - (windowSize / 2) - 1;

	// allocate memory
	float* currMsg;
	hipMallocManaged(&currMsg, 4*numLabels*numPixels*sizeof(float));
	currMsg = new float[4*numLabels*numPixels];

	float* prevMsg;
	hipMallocManaged(&prevMsg, 4*numLabels*numPixels*sizeof(float));
	prevMsg = new float[4*numLabels*numPixels];

	float* belief;
	hipMallocManaged(&belief, numLabels*numPixels*sizeof(float));
	belief = new float[numLabels*numPixels];

	float* DC;
	hipMallocManaged(&DC, numLabels*numPixels*sizeof(float));
	DC = new float[numLabels*numPixels];

	// memoize data costs
	for (int x = leftBorder; x <= rightBorder; x++) {
		for (int y = topBorder; y <= bottomBorder; y++) {
			for (int k = 0; k < numLabels; k++) {
				int sum = 0;
				int absDiff;
				for (int i = x - windowSize / 2; i <= x + windowSize / 2; i++) {
					for (int j = y - windowSize / 2; j <= y + windowSize / 2; j++) {
						absDiff = abs((int) leftPixels[j*width+i]
							- (int) rightPixels[(j-k)*width+i]);
						sum += absDiff;
					}
				}
				setDC(x, y, k, (float) sum, &DC);
			}
		}
	}

    // print out all DC vectors for debugging
	for (int x = leftBorder; x <= rightBorder; x++) {
		for (int y = topBorder; y <= bottomBorder; y++) {
			if (x%100==0 && y%100==0) {
				std::cout << "(" << x << "," << y << ")\t[";
				for (int k = 0; k < numLabels; k++) {
					std::cout << getDC(x, y, k, &DC) << ", ";
				}//End of for
				std::cout << "]\n";
			}//End of if
		}//End of for
	}//End of for

	// set up buffers
	int* inBufferData;

	//Would I need to create another inBufferData variable? 
	//Would I need to create another InputBuffer?
	//If how would I deal with therons calculations (updated belief) output to use for my input of my calcualtions (energy calcuations)

	hipMallocManaged(&inBufferData, numPixels*sizeof(int));

	Mercator::InputBuffer<int>* inBuffer = new Mercator::InputBuffer<int>(inBufferData, numPixels);

	Mercator::OutputBuffer<int>* outBuffer = new Mercator::OutputBuffer<int>(numPixels);

	Mercator::OutputBuffer<int>∗ outBuffer2 = new Mercator::OutputBuffer<int >(OUT_BUFFER_CAPACITY1); //How to know how much of a out buffer capacity I need?
	//do I say inBufferData?

	for (int x = leftBorder; x <= rightBorder; x++) {
		for (int y = topBorder; y <= bottomBorder; y++) {
			inBuffer->add(y*width+x);
		}
	}

	// create app object
	UpdateApp* updateApp = new UpdateApp();

	//create energyApp object
	EnergyApp* energyApp = new EnergyApp();

	// set app-level data
	StereoVisionAppData* appData = new StereoVisionAppData(
		currMsg, prevMsg,
		belief,
		DC,
		numLabels,
		lambda, truncate,
		width, height
		);
	updateApp->set_userData(appData);
	energyApp-> set_userData(energyApp);

	// associate buffers with nodes

	updateApp->sourceNode->set_inBuffer(inBuffer);
	updateApp->sinkNode->set_outBuffer(outBuffer);

	updateApp->run();
	energyApp->run();

	std::cout << "App run completed" << endl;

//   synthApp0->run();

//   std::cout << "SynthApp0 finished.\n" ;

//   // print contents of output buffer
// #if PRINT_OUTPUT_BUFFERS
//   std::cout << " Output buffers: \n" ;

//   int* outData1 = outBuffer1->get_data();
//   printf("SynthApp0, OutBuffer1 (%p):\n", outBuffer1);
//   for(int i=0; i < outBuffer1->size(); ++i)
//   	printf("[%d]: %d\n", i, outData1[i]);

//   int* outData2 = outBuffer2->get_data();
//   printf("SynthApp0, OutBuffer2 (%p):\n", outBuffer2);
//   for(int i=0; i < outBuffer2->size(); ++i)
//   	printf("[%d]: %d\n", i, outData2[i]);

//   int* outData3 = outBuffer3->get_data();
//   printf("SynthApp0, OutBuffer3 (%p):\n", outBuffer3);
//   for(int i=0; i < outBuffer3->size(); ++i)
//   	printf("[%d]: %d\n", i, outData3[i]);
// #endif   // print contents of output buffer

//   // cleanup
//   hipFree(inBufferData);
//   hipFree(synthApp0);

}

void importPGM(string filePath, unsigned char** pixels) {
	string line;
	ifstream file(filePath.c_str());
	if (file.is_open()) {
		getline(file, line);
		line = line.substr(0, 2);
		if (line.compare("P2") != 0) {
			std::cout << "Incorrect file type" << endl;
			exit(EXIT_FAILURE);
		}
		getline(file, line);
		stringstream ss(line);
		char c;
		int ifWidth, ifHeight;
		ss >> c;
        if (c == '#') { // ignore this line of the file
        	getline(file, line);
        	ss.str(line);
        	ss >> ifWidth >> ifHeight;
        }
        else { // that char is actually the width
        	ifWidth = (int) c;
        	ss >> ifHeight;
        }
        width = ifWidth;
        height = ifHeight;
        std::cout << "Width: " << width << " Height: " << height << endl;
        numPixels = width * height;
        std::cout << "numPixels: " << numPixels << endl;
        *pixels = new unsigned char[numPixels];
        getline(file, line); // read maximum gray value
        int pixel;
        int count = 0;
        while (!file.eof()) {
        	getline(file, line);
        	ss.clear();
        	ss.str(line);
        	while (ss >> pixel) {
        		(*pixels)[count] = pixel;
        		count++;
        	}
        }
        std::cout << "Import successful" << endl << endl;
    }
    else {
    	std::cout << "Import failed" << endl;
    	exit(EXIT_FAILURE);
    }
}
void setDC(int x, int y, int label, float val, float** DCArray) {
	(*DCArray)[(numPixels*label) + (y*width+x)] = val;
}
float getDC(int x, int y, int label, float** DCArray) {
	return (*DCArray)[(numPixels*label) + (y*width+x)];
}
#endif